#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void blur_kernel(int *A,int r,int c)
{
	int row=blockIdx.x*blockDim.x + threadIdx.x
	int col=blockIdx.y*blockDim*y + threadIdx.y

	if(row<r&&col<c)
	{
		A[row*c+col]*=5;
	}
}

int main()
{
	int dimx=5;
	int dimy=4;

	int *A,*res;
	A=(int*)malloc(sizeof(int)*dimx*dimy);
	res=(int*)malloc(sizeof(int)*dimx*dimy);

	for(int i=0;i<5;i++)
	{
		for(int j=0;j<4;j++)
		{
			A[i*4+j]=1;
		}
	}

	int *d_a;
	hipMalloc((void**)&d_a,sizeof(int)*dimx*dimy);

	hipMemcpy(d_a,A,sizeof(int)*dimx*dimy,hipMemcpyHostToDevice);

	dim3 gridDim(3,2);
	dim3 blockDim(2,2);

	blur_kernel<<<gridDim,blockDim>>>(d_a,dimx,dimy);

	hipMemcpy(res,d_a,sizeof(int)*dimx*dimy,hipMemcpyDeviceToHost);

	for(int i=0;i<5;i++)
	{
		for(int j=0;j<4;j++)
		{
			printf("%d ",res[i*4+j]);
		}
		printf("\n");
	}
	printf("\n");
	free(A);
	free(res);
	hipFree(d_a);

	return 0;
}