
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void blur_kernel(float *image,float *filter,float *blurred,int r,int c,float filter_sum)
{
        int row=blockIdx.x*blockDim.x + threadIdx.x;
        int col=blockIdx.y*blockDim.y + threadIdx.y;
        int above=row-1;
        int below=row+1;
        int left=col-1;
        int right=col+1;
        int total_sum=0;

        // Checking conditions for each pixel value
        if(0<=above&&above<r){
                if(0<=left&&left<c)
                        total_sum+=(image[above*c+left]*filter[0*3+0]);
                if(0<=col&&col<c)
                        total_sum+=(image[above*c+col]*filter[0*3+1]);
                if(0<=right&&right<c)
                        total_sum+=(image[above*c+right]*filter[0*3+2]);
        }

        if(0<=row&&row<r){
                if(0<=left&&left<c)
                        total_sum+=(image[row*c+left]*filter[1*3+0]);
                if(0<=col&&col<c)
                        total_sum+=(image[row*c+col]*filter[1*3+1]);
                if(0<=right&&right<c)
                        total_sum+=(image[row*c+right]*filter[1*3+2]);
        }

        if(0<=below&&below<r){
                if(0<=left&&left<c)
                        total_sum+=(image[below*c+left]*filter[2*3+0]);
                if(0<=col&&col<c)
                        total_sum+=(image[below*c+col]*filter[2*3+1]);
                if(0<=right&&right<c)
                        total_sum+=(image[below*c+right]*filter[2*3+2]);
        }
        if((0<=row&&row<r)&&(0<=col&&col<c))
                blurred[row*c+col]=total_sum/filter_sum;
}

int main()
{
        srand(time(NULL));

        // Image of size 1000*768
        int dimx=1000;
        int dimy=768;

        // Device variables and memory allocation
        float *image,*filter,*blurred;
        image=(float*)malloc(sizeof(float)*dimx*dimy);
        filter=(float*)malloc(sizeof(float)*3*3);
        blurred=(float*)malloc(sizeof(float)*dimx*dimy);

        // Assigning fixed values to the filter matrix
        filter[0*3+0]=1; filter[0*3+1]=2; filter[0*3+2]=1;
        filter[1*3+0]=2; filter[1*3+1]=3; filter[1*3+2]=2;
        filter[2*3+0]=1; filter[2*3+1]=2; filter[2*3+2]=1;

        // Assigning random values between 0 and 255 to the image matrix
        for(int i=0;i<dimx;i++)
        {
                for(int j=0;j<dimy;j++)
                {
                        image[i*dimy+j]=rand()%256;
                }
        }

        // Device variables and memory allocation
        float *d_image,*d_filter,*d_blurred;
        hipMalloc((void**)&d_image,sizeof(float)*dimx*dimy);
        hipMalloc((void**)&d_filter,sizeof(float)*3*3);
        hipMalloc((void**)&d_blurred,sizeof(float)*dimx*dimy);

        // Copying the image and filter matrix to device memory
        hipMemcpy(d_image,image,sizeof(float)*dimx*dimy,hipMemcpyHostToDevice);
        hipMemcpy(d_filter,filter,sizeof(float)*3*3,hipMemcpyHostToDevice);

        // Grid and block dimensions
        dim3 gridDim(63,48);
        dim3 blockDim(16,16,1);

        // Calling the kernel
        blur_kernel<<<gridDim,blockDim>>>(d_image,d_filter,d_blurred,dimx,dimy,15.0);

        // Copying the blurred image from device to host
        hipMemcpy(blurred,d_blurred,sizeof(float)*dimx*dimy,hipMemcpyDeviceToHost);

        // Printing original image
        printf("Original Image\n");
        for(int i=0;i<dimx;i++)
        {
                for(int j=0;j<dimy;j++)
                {
                        printf("%f ",image[i*dimy+j]);
                }
                printf("\n");
        }

        // Printing blurred image
        printf("\n\nBlurred Image\n");
        for(int i=0;i<dimx;i++)
        {
                for(int j=0;j<dimy;j++)
                {
                        printf("%f ",blurred[i*dimy+j]);
                }
                printf("\n");
        }

        // Freeing up memory
        free(image);
        free(filter);
        free(blurred);
        hipFree(d_image);
        hipFree(d_filter);
        hipFree(d_blurred);

        return 0;
}                                                                                                                       